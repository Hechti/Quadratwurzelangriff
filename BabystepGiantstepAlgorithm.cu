#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "InfInt.h"
#include "Utilities.h"
#include <map>
#include "BabystepGiantstepAlgorithm.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

void printBabyStepTable(std::map<InfInt, InfInt> mapBabyStep)
{
    
	printf("\tBabyStep j [");
    for (auto &value : mapBabyStep)
    {
        printf("%s,", value.first.toString().c_str());
    }
	printf("\b]\n");
}

__global__ void baby(const unsigned int *m, const ll *g, const ll *n, const unsigned int *offset, ll *babyStepTable)
{
    // ID berechnen
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lowerLimit;
    unsigned int higherLimit;

    // untere und obere Grenze bestimmen
    lowerLimit = id * *offset;
    higherLimit = lowerLimit + *offset;

    // J-Tabelle berechnen mit: g^j mod n
    for (unsigned int j = lowerLimit; j < higherLimit && j < *m; j++)
    {
        cudaPowModll(g, (ll*) &j, n, &babyStepTable[j]);
    }
}

typedef struct
{
    unsigned int i;
    unsigned int j;
} CudaResult;

__global__ void giant(const unsigned int *m, const ll *g, const ll *n, const ll *a, const unsigned int *offset, const ll *babyStepTable, CudaResult *result, int *isResultFound)
{
    // ID berechnen
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lowerLimit;
    unsigned int higherLimit;

    // untere und obere Grenze bestimmen
    lowerLimit = id * *offset;
    higherLimit = lowerLimit + *offset;

    // Jede GPU arbeitet ihren Block ab, auszer es wurde ein Ergebnis gefunden
    for (unsigned int i = lowerLimit; i < higherLimit && i < *m && !*isResultFound; i++)
    {
        ll exp = *m;
        exp -= *n;
        exp = (exp -1) * i;
        
        ll tmpResult = 0;
        cudaPowModll(g, &exp, n, &tmpResult);
        tmpResult *= *a;

        for (unsigned int j = 0; j < *m && !*isResultFound; j++)
        {
            if (tmpResult == babyStepTable[j])
            {
                // Atomares zuweisen notwendig, da es vorkommen kann, 
                // dass mehrere gueltige Ergebnisse gefunden werden
                atomicAdd(isResultFound, 1);
                atomicAdd(&(result->j), babyStepTable[j]);
                atomicAdd(&(result->i), i);

                return;
            }
        }
    }
}

void babyGiant(const InfInt& n, const InfInt& g, const InfInt& a, ll &result)
{
	const unsigned int MAX_BLOCK_SIZE = 65536;
    const unsigned int MAX_THREAD_SIZE = 1024;
    unsigned int m = ((n-1).intSqrt() + 1).toUnsignedInt();
    
    unsigned int numberOfBlocks;
    unsigned int numberOfThreads = 1;
    unsigned int offset = 0;

    // Berechnung der Anzahl der benoetigten Threads und einem offset, 
    // da unter umstaenden jeder CUDA-Core mehrere Berechnungen durchfuehren muss
    if (m >= 65536)
    {
        numberOfBlocks = MAX_BLOCK_SIZE;
        numberOfThreads = (m / MAX_BLOCK_SIZE) + 1;

        if (numberOfThreads >= 1024)
        {
            offset = (m / (MAX_BLOCK_SIZE * MAX_THREAD_SIZE)) + 1;
        }
    }
    else
    {
        numberOfBlocks = m;
    }

    // Deklaration aller CUDA-Variablen
    // ll *hostBabyStepTable; 
    ll *deviceBabyStepTable;
    unsigned int *deviceM;
    ll *deviceN;
    ll *deviceG;
    ll *deviceA;
    unsigned int *deviceOffset;
    CudaResult *deviceResult;
    int isResultFound = 0;
    int *deviceIsResultFound;

    // Allokiern von Grafikartenspeicher
    CHECK(hipMalloc((void**) &deviceM, sizeof(unsigned int)));
    CHECK(hipMalloc((void**) &deviceN, sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceG, sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceA, sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceOffset, sizeof(unsigned int)));
    CHECK(hipMalloc((void**) &deviceBabyStepTable, m * sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceResult, sizeof(CudaResult)));
    CHECK(hipMalloc((void**) &deviceIsResultFound, sizeof(int)));

    // Daten auf die Grafikkarte kopieren
    CHECK(hipMemcpy(deviceM, &m, sizeof(unsigned int), hipMemcpyHostToDevice));
    ll value = n.toUnsignedLongLong();
    CHECK(hipMemcpy(deviceN, &value, sizeof(ll), hipMemcpyHostToDevice));
    value = g.toUnsignedLongLong();
    CHECK(hipMemcpy(deviceG, &value, sizeof(ll), hipMemcpyHostToDevice));
    value = a.toUnsignedLongLong();
    CHECK(hipMemcpy(deviceA, &value, sizeof(ll), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(deviceOffset, &offset, sizeof(unsigned int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(deviceIsResultFound, &isResultFound, sizeof(int), hipMemcpyHostToDevice));

    baby<<<numberOfBlocks, numberOfThreads>>>(deviceM, deviceG, deviceN, deviceOffset, deviceBabyStepTable);
    giant<<<numberOfBlocks, numberOfThreads>>>(deviceM, deviceG, deviceN, deviceA, deviceOffset, deviceBabyStepTable, deviceResult, deviceIsResultFound);

    CudaResult hostResult;
    CHECK(hipMemcpy(&hostResult, deviceResult, sizeof(CudaResult), hipMemcpyDeviceToHost));

    printf("i: %u, j: %u\n", hostResult.i, hostResult.j);

    // Grafikkartenspeicher freigeben
    CHECK(hipFree(deviceM));
    CHECK(hipFree(deviceN));
    CHECK(hipFree(deviceG));
    CHECK(hipFree(deviceA));
    CHECK(hipFree(deviceOffset));
    CHECK(hipFree(deviceBabyStepTable));
    CHECK(hipFree(deviceResult));
    CHECK(hipFree(deviceIsResultFound));
}

void babystepGiantstepAlgorithm(const InfInt& n, const InfInt& g, const InfInt& a, InfInt &secretResult)
{
	InfInt m = (n-1).intSqrt() + 1;
    printf("\tm: %s\n", m.toString().c_str());
    
    std::map<InfInt, InfInt> mapBabyStep;
	for (InfInt j=0; j<m; j++)
	{
		InfInt result;
		powModulo(g, j, n, result);
		mapBabyStep[result] = j;
	}

    if (m < InfInt(100))
    {
        printBabyStepTable(mapBabyStep);
    }

	for (InfInt i=0; i<m; i++)
	{
		// InfInt exp = (n - 1) - (i * m);
        InfInt one = 1;
        InfInt exp = (n - m - one) * i;
		InfInt tmpErg; 
		powModulo(g, exp, n, tmpErg);
		InfInt result = (a * tmpErg) % n;
		
        auto it = mapBabyStep.find(result);
        if (it != mapBabyStep.end())
        {
            secretResult = i * m + it->second;
        	printf("\tsecret result: [%s]\n\n", secretResult.toString().c_str());
            return;
        }

	}
}

void babystepGiantstepAlgorithmCUDA(const InfInt &n, const InfInt &g, const InfInt &a, InfInt &result)
{
	const unsigned int BABY_TABLE_COLOUMN_SIZE = 65536;
    unsigned int m = ((n - 1).intSqrt() + 1).toUnsignedInt();
    
    printf("m: %u\n", m);

    unsigned int babyTableRowSize;
    ll **babyTable;
    if (m < BABY_TABLE_COLOUMN_SIZE)
    {
        babyTable = new ll*[1];
        babyTable[0] = new ll[m];
        
        babyTableRowSize = 1;
    }
    else
    {
        babyTableRowSize = m / BABY_TABLE_COLOUMN_SIZE;
        babyTableRowSize += 1;

        babyTable = new ll*[babyTableRowSize];

        for (int i = 0; i < babyTableRowSize; i++)
        {
            babyTable[i] = new ll[BABY_TABLE_COLOUMN_SIZE];
        }   
    }



    ll *mapBabyStep = (ll*)malloc(m * sizeof(ll));
    ll *deviceN, *deviceG, *deviceMapBabyStep;
    unsigned int *deviceM, *deviceOffset;
    
    hipMalloc((void**) &deviceN, sizeof(ll));
    hipMalloc((void**) &deviceM, sizeof(unsigned int));
    hipMalloc((void**) &deviceG, sizeof(ll));
    hipMalloc((void**) &deviceOffset, sizeof(unsigned int));

    ll value = n.toUnsignedLongLong();
    hipMemcpy(deviceN, &value, sizeof(ll), hipMemcpyHostToDevice);
    value = g.toUnsignedLongLong();
    hipMemcpy(deviceG, &value, sizeof(ll), hipMemcpyHostToDevice);
    // hipMemcpy(deviceMapBabyStep, mapBabyStep, m * sizeof(ll), hipMemcpyHostToDevice);
   
    if (babyTableRowSize == 1)
    {
        value = 0;
        hipMemcpy(deviceOffset, &value, sizeof(unsigned int), hipMemcpyHostToDevice);
        
        hipMemcpy(deviceM, &m, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMalloc((void**) &deviceMapBabyStep, m * sizeof(ll));
        
        babyStep<<<m, 1>>>(deviceN, deviceM, deviceG, deviceOffset, deviceMapBabyStep);
        hipMemcpy(babyTable[0], deviceMapBabyStep, m * sizeof(ll), hipMemcpyDeviceToHost);
    }
    else
    {
        hipMalloc((void**) &deviceMapBabyStep, BABY_TABLE_COLOUMN_SIZE * sizeof(ll));
        hipMemcpy(deviceM, &BABY_TABLE_COLOUMN_SIZE, sizeof(unsigned int), hipMemcpyHostToDevice);
        
        for (unsigned int i = 0; i < babyTableRowSize - 1; i++)
        {
            hipMemcpy(deviceOffset, &i, sizeof(unsigned int), hipMemcpyHostToDevice);
            babyStep<<<BABY_TABLE_COLOUMN_SIZE, 1>>>(deviceN, deviceM, deviceG, deviceOffset, deviceMapBabyStep);
            hipMemcpy(babyTable[i], deviceMapBabyStep, BABY_TABLE_COLOUMN_SIZE * sizeof(ll), hipMemcpyDeviceToHost);
        }
        
        hipMemcpy(deviceM, &m, sizeof(unsigned int), hipMemcpyHostToDevice);
        unsigned int offset = babyTableRowSize - 1;
        hipMemcpy(deviceOffset, &offset, sizeof(unsigned int), hipMemcpyHostToDevice);
        
        babyStep<<<m, 1>>>(deviceN, deviceM, deviceG, deviceOffset, deviceMapBabyStep);
        hipMemcpy(babyTable[babyTableRowSize - 1], deviceMapBabyStep, m * sizeof(ll), hipMemcpyDeviceToHost);
    }

    if (m <= 100)
    {
        printf("[");
        for (int i = 0; i < babyTableRowSize - 1; i++)
        {
            for (int j = 0; j < BABY_TABLE_COLOUMN_SIZE - 1; j++)
            {
                printf("%llu,", babyTable[i][j]);
            }
        }
        for (int j = 0; j < m; j++)
        {
            printf("%llu,", babyTable[babyTableRowSize - 1][j]);
        }
        printf("\b]\n\n");
    }

    hipFree(deviceN);
    hipFree(deviceM);
    hipFree(deviceG);
    hipFree(deviceMapBabyStep);
    
    for (int i = 0; i < babyTableRowSize; i++) 
    {
        delete [] babyTable[i];
    }

    delete [] babyTable;

    free(mapBabyStep);
}

__global__ void babyStep(const ll *n, const unsigned int *m, const ll *g, const unsigned int *offset, ll *mapBabyStep) 
{
	const unsigned int BABY_TABLE_COLOUMN_SIZE = 65536;
    ll id = blockIdx.x + (BABY_TABLE_COLOUMN_SIZE * *offset);
    cudaPowModll(g, &id, n, &mapBabyStep[id]);
}

__global__ void giantStep(const ll *n, const unsigned int *m, const ll *g, const ll *a, ll *mapBabyStep, unsigned int *resultI, unsigned int *resultJ, int *foundResult)
{
    if (!foundResult)
    {
        ll id = blockIdx.x;
        ll localN, localM;
        localN = *n;
        localM = *m;
        ll exp = (localN - localM - 1) * id;
        ll powResult;
        cudaPowModll(g, &exp, n, &powResult);
        powResult = (powResult * *a) % *n;

        for (unsigned int i = 0; i < *m; i++)
        {
                if (mapBabyStep[i] == powResult && !foundResult)
                {
                    atomicAdd(foundResult, 1);
                    atomicAdd(resultJ, i);
                    atomicAdd(resultI, id);
                    return;
                }
        }
    }
}
typedef struct
{
    ll key;
    ll data;
} CudaPowData;

__device__ void cudaPow(const ll *basis, const ll *exponent, const ll *modulus, ll *result)
{
    ll check1 = 0;
    ll check2 = 1;

    if (*basis == check1)
    {
        *result = check1;
        return;
    }

    if (*exponent == check1)
    {
        *result = check2;
        return;
    }

    if (*exponent == check2)
    {
        *result = *basis;
        return;
    }

    int arraySize = 0;
    int arrayCount = 0;
    getArraySize(exponent, &arraySize);
    
    CudaPowData *values = new CudaPowData[arraySize];
    
    ll globalExp = 1;
    *result = *basis;

    do
    {
        if ((globalExp + globalExp) <= *exponent)
        {
            *result *= *result;
            *result %= *modulus;
            globalExp *= 2;

            CudaPowData data;
            data.key = globalExp;
            data.data = *result;

            values[arrayCount] = data;
        }
        else
        {
            if ((*exponent - globalExp) == 1)
            {
                *result *= *basis;
                *result %= *modulus;
                globalExp += 1;
            }
            else
            {
                for (int i = arraySize - 1; i >= 0; i--)
                {
                    if ((values[i].key + globalExp) <= *exponent)
                    {
                        *result *= values[i].data;
                        *result %= *modulus;
                        globalExp += values[i].key;
                    }
                }
            }
        }
    } while (globalExp != *exponent);

    delete [] values;
}

__device__ void getArraySize(const ll *exp, int *arraySize)
{
    *arraySize = 0;
    ll value = 1;

    while (value <= *exp)
    {
        value *= 2;
        *arraySize++;
    } 
    *arraySize -= 1;
}

__device__ void cudaPowModll(const ll* base, const ll* exp, const ll* mod, ll* result)
{
	if (*exp == 0)
	{
		*result = 1;
		return;
	}

	int i;
	for (i = 62; i>=1; --i)
	{
		if (((*exp>>i)&1) == 1)
		{
			break;
		}
	}
	*result = *base;
	for (--i; i >=0; --i)
	{
		*result *= *result;
		*result %= *mod;
		if ((*exp>>i)&1)
		{
			*result *= *base;
			*result %= *mod;
		}
	}
}
