#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "Lock.h"
#include "InfInt.h"
#include "Utilities.h"
#include "DiffieHellman.h"
#include <map>
#include "BabystepGiantstepAlgorithm.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

void printBabyStepTable(std::map<InfInt, InfInt> mapBabyStep)
{
    
	printf("\tBabyStep j [");
    for (auto &value : mapBabyStep)
    {
        printf("%s,", value.first.toString().c_str());
    }
	printf("\b]\n");
}

__global__ void baby(const unsigned int *m, const ll *g, const ll *n, const unsigned int *offset, ll *babyStepTable)
{
    // ID berechnen
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lowerLimit;
    unsigned int higherLimit;

    // untere und obere Grenze bestimmen
    lowerLimit = id * *offset;
    higherLimit = lowerLimit + *offset;

    // J-Tabelle berechnen mit: g^j mod n
    for (unsigned int j = lowerLimit; j < higherLimit && j < *m; j++)
    {
        cudaPowModll(g, (ll*) &j, n, &babyStepTable[j]);
    }
}

__device__ size_t highestOneBitPosition(ll a) 
{
    size_t bits = 0;
    while (a != 0) {
        ++bits;
        a >>= 1;
    };

    return bits;
}

__device__ bool isMultiplicationSafe(ll a, ll b)
{
    size_t a_bits = highestOneBitPosition(a);
    size_t b_bits = highestOneBitPosition(b);
    
    return (a_bits + b_bits <= 64);
}

__global__ void giant(const unsigned int *m, const ll *g, const ll *n, const ll *a, const unsigned int *offset, const ll *babyStepTable, CudaResult *result, Lock lock)
{
    // ID berechnen
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lowerLimit;
    unsigned int higherLimit;
    
    // create a shared variable and initialize
    __shared__  int isResultFound;
    lock.foundResult(isResultFound);

    // untere und obere Grenze bestimmen
    lowerLimit = id * *offset;
    higherLimit = lowerLimit + *offset;
    // printf("ID: %u, offset: %u, UG: %u, OG: %u\n", id, *offset, lowerLimit, higherLimit);

    // Jede GPU arbeitet ihren Block ab, auszer es wurde ein Ergebnis gefunden
    for (unsigned int i = lowerLimit; i < higherLimit && i < *m && !isResultFound; i++)
    {
        ll exp = *n;
        exp -= *m;
        exp = (exp -1) * i;
        
        ll tmpResult = 0;
        cudaPowModll(g, &exp, n, &tmpResult);
        // tmpResult *= *a;

        if (!isMultiplicationSafe(tmpResult, *a))
        {
            printf("overflow detected\n");
        }
        else
        {
            tmpResult *= *a;
        }

        tmpResult %= *n;
        // printf("g ** exp mod n = %llu ** %llu mod %llu = %llu\n", *g, exp, *n, tmpResult);

        for (unsigned int j = 0; j < *m && !isResultFound; j++)
        {
            if (tmpResult == babyStepTable[j])
            {
                // Atomares zuweisen notwendig, da es vorkommen kann, 
                // dass mehrere gueltige Ergebnisse gefunden werden
                // while(atomicCAS(mutex, 0, 1) != 0);
                lock.lock();
                lock.foundResult(isResultFound);

                if (!isResultFound)
                {
                    result->j = j;
                    result->i = i;
                    lock.setFoundResult(isResultFound);
                    isResultFound = true;

                    printf("found result: (%u, %u) -> %llu\n", i, j, tmpResult);
                }
                // atomicExch(mutex, 0);
                lock.unlock();

                return;
            }
        }
    }
}

void babyGiant(InfInt &n, InfInt &g, InfInt &a, InfInt &b, InfInt &result)
{
	const unsigned int MAX_BLOCK_SIZE = 65535;
    const unsigned int MAX_THREAD_SIZE = 1023;
    unsigned int m = ((n-1).intSqrt() + 1).toUnsignedInt();
    
    unsigned int numberOfBlocks;
    unsigned int numberOfThreads = 1;
    unsigned int offset = 1;

    // Berechnung der Anzahl der benoetigten Threads und einem offset, 
    // da unter umstaenden jeder CUDA-Core mehrere Berechnungen durchfuehren muss
    if (m > MAX_BLOCK_SIZE)
    {
        numberOfBlocks = MAX_BLOCK_SIZE;
        numberOfThreads = (m / MAX_BLOCK_SIZE) + 1;

        if (numberOfThreads >= MAX_THREAD_SIZE)
        {
            numberOfThreads = MAX_THREAD_SIZE;
            offset = (m / (MAX_BLOCK_SIZE * MAX_THREAD_SIZE)) + 1;
        }
    }
    else
    {
        numberOfBlocks = m;
    }

    printf("\n\nStartet CUDA with %u blocks, %u threads, offset = %u, and m = %u!\n\n", numberOfBlocks, numberOfThreads, offset, m);

    // Deklaration aller CUDA-Variablen
    ll *hostBabyStepTable; 
    ll *deviceBabyStepTable;
    unsigned int *deviceM;
    ll *deviceN;
    ll *deviceG;
    ll *deviceA;
    ll *deviceB;
    unsigned int *deviceOffset;
    CudaResult hostResultAlice;
    CudaResult *deviceResultAlice;
    CudaResult hostResultBob;
    CudaResult *deviceResultBob;

    // DEBUG
    hostBabyStepTable = new ll[m];

    // Allokiern von Grafikartenspeicher
    CHECK(hipMalloc((void**) &deviceM, sizeof(unsigned int)));
    CHECK(hipMalloc((void**) &deviceN, sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceG, sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceA, sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceB, sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceOffset, sizeof(unsigned int)));
    CHECK(hipMalloc((void**) &deviceBabyStepTable, m * sizeof(ll)));
    CHECK(hipMalloc((void**) &deviceResultAlice, sizeof(CudaResult)));
    CHECK(hipMalloc((void**) &deviceResultBob, sizeof(CudaResult)));

    // Daten auf die Grafikkarte kopieren
    CHECK(hipMemcpy(deviceM, &m, sizeof(unsigned int), hipMemcpyHostToDevice));
    
    ll value = n.toUnsignedLongLong();
    CHECK(hipMemcpy(deviceN, &value, sizeof(ll), hipMemcpyHostToDevice));
    
    value = g.toUnsignedLongLong();
    CHECK(hipMemcpy(deviceG, &value, sizeof(ll), hipMemcpyHostToDevice));
    
    value = a.toUnsignedLongLong();
    CHECK(hipMemcpy(deviceA, &value, sizeof(ll), hipMemcpyHostToDevice));

    value = b.toUnsignedLongLong();
    CHECK(hipMemcpy(deviceB, &value, sizeof(ll), hipMemcpyHostToDevice));

    CHECK(hipMemcpy(deviceOffset, &offset, sizeof(unsigned int), hipMemcpyHostToDevice));

    hostResultAlice.i = 0;
    hostResultAlice.j = 0;
    CHECK(hipMemcpy(deviceResultAlice, &hostResultAlice, sizeof(CudaResult), hipMemcpyHostToDevice));
    
    hostResultBob.i = 0;
    hostResultBob.j = 0;
    CHECK(hipMemcpy(deviceResultBob, &hostResultBob, sizeof(CudaResult), hipMemcpyHostToDevice));

    // Fuelle die BabStep Tabelle
    baby<<<numberOfBlocks, numberOfThreads>>>(deviceM, deviceG, deviceN, deviceOffset, deviceBabyStepTable);

    // DEBUG
    CHECK(hipMemcpy(hostBabyStepTable, deviceBabyStepTable, m * sizeof(ll), hipMemcpyDeviceToHost));

    if (m < 100)
    {
        printf("Table j: [");
        for (unsigned int i = 0; i < m; i++)
        {
            printf("%llu,", hostBabyStepTable[i]);
        }
        printf("\b]\n\n");

    }

    // Suche nach Alice's Eingabe
    Lock lockA;
    giant<<<numberOfBlocks, numberOfThreads>>>(deviceM, deviceG, deviceN, deviceA, deviceOffset, deviceBabyStepTable, deviceResultAlice, lockA);

    // Suche nach Bob's Eingabe
    Lock lockB;
    giant<<<numberOfBlocks, numberOfThreads>>>(deviceM, deviceG, deviceN, deviceB, deviceOffset, deviceBabyStepTable, deviceResultBob, lockB);

    // Ausgabe Ergebnis Alice
    CHECK(hipMemcpy(&hostResultAlice, deviceResultAlice, sizeof(CudaResult), hipMemcpyDeviceToHost));
    printf("\nAlice:\n");
    printf("i: %u, j: %u\n", hostResultAlice.i, hostResultAlice.j);
    InfInt ergAlice = (InfInt(hostResultAlice.i) * InfInt(m)) + InfInt(hostResultAlice.j);
    printf("Ergebnis: %s\n", ergAlice.toString().c_str());

    // Ausgabe Ergebnis Bob
    CHECK(hipMemcpy(&hostResultBob, deviceResultBob, sizeof(CudaResult), hipMemcpyDeviceToHost));
    printf("\nBob:\n");
    printf("i: %u, j: %u\n", hostResultBob.i, hostResultBob.j);
    InfInt ergBob = (InfInt(hostResultBob.i) * InfInt(m)) + InfInt(hostResultBob.j);
    printf("Ergebnis: %s\n", ergBob.toString().c_str());

    InfInt alice(ergAlice);
    InfInt bob(ergBob);
    InfInt pseudo1, pseudo2;
    diffieHellman(n, g, alice, bob, pseudo1, pseudo2, result);

    printf("\n\ncalculated private key: %s\n\n", result.toString().c_str());


    // DEBUG
    delete [] hostBabyStepTable;
    // Grafikkartenspeicher freigeben
    CHECK(hipFree(deviceM));
    CHECK(hipFree(deviceN));
    CHECK(hipFree(deviceG));
    CHECK(hipFree(deviceA));
    CHECK(hipFree(deviceB));
    CHECK(hipFree(deviceOffset));
    CHECK(hipFree(deviceBabyStepTable));
    CHECK(hipFree(deviceResultAlice));
    CHECK(hipFree(deviceResultBob));
}

void babystepGiantstepAlgorithm(const InfInt& n, const InfInt& g, const InfInt& a, InfInt &secretResult)
{
	InfInt m = (n-1).intSqrt() + 1;
    printf("\tm: %s\n", m.toString().c_str());
    
    std::map<InfInt, InfInt> mapBabyStep;
	for (InfInt j=0; j<m; j++)
	{
		InfInt result;
		powModulo(g, j, n, result);
		mapBabyStep[result] = j;
	}

    if (m < InfInt(100))
    {
        printBabyStepTable(mapBabyStep);
    }

	for (InfInt i=0; i<m; i++)
	{
		// InfInt exp = (n - 1) - (i * m);
        InfInt one = 1;
        InfInt exp = (n - m - one) * i;
		InfInt tmpErg; 
		powModulo(g, exp, n, tmpErg);
		InfInt result = (a * tmpErg) % n;
        // printf("g ** exp mod n = %llu ** %llu mod %llu = %llu\n", g.toUnsignedLongLong(), exp.toUnsignedLongLong(), n.toUnsignedLongLong(), result.toUnsignedLongLong());
		
        auto it = mapBabyStep.find(result);
        if (it != mapBabyStep.end())
        {
            secretResult = i * m + it->second;
        	printf("\tsecret result: [%s]\n\n", secretResult.toString().c_str());
            return;
        }

	}
}

void babystepGiantstepAlgorithmCUDA(const InfInt &n, const InfInt &g, const InfInt &a, InfInt &result)
{
	const unsigned int BABY_TABLE_COLOUMN_SIZE = 65536;
    unsigned int m = ((n - 1).intSqrt() + 1).toUnsignedInt();
    
    printf("m: %u\n", m);

    unsigned int babyTableRowSize;
    ll **babyTable;
    if (m < BABY_TABLE_COLOUMN_SIZE)
    {
        babyTable = new ll*[1];
        babyTable[0] = new ll[m];
        
        babyTableRowSize = 1;
    }
    else
    {
        babyTableRowSize = m / BABY_TABLE_COLOUMN_SIZE;
        babyTableRowSize += 1;

        babyTable = new ll*[babyTableRowSize];

        for (int i = 0; i < babyTableRowSize; i++)
        {
            babyTable[i] = new ll[BABY_TABLE_COLOUMN_SIZE];
        }   
    }



    ll *mapBabyStep = (ll*)malloc(m * sizeof(ll));
    ll *deviceN, *deviceG, *deviceMapBabyStep;
    unsigned int *deviceM, *deviceOffset;
    
    hipMalloc((void**) &deviceN, sizeof(ll));
    hipMalloc((void**) &deviceM, sizeof(unsigned int));
    hipMalloc((void**) &deviceG, sizeof(ll));
    hipMalloc((void**) &deviceOffset, sizeof(unsigned int));

    ll value = n.toUnsignedLongLong();
    hipMemcpy(deviceN, &value, sizeof(ll), hipMemcpyHostToDevice);
    value = g.toUnsignedLongLong();
    hipMemcpy(deviceG, &value, sizeof(ll), hipMemcpyHostToDevice);
    // hipMemcpy(deviceMapBabyStep, mapBabyStep, m * sizeof(ll), hipMemcpyHostToDevice);
   
    if (babyTableRowSize == 1)
    {
        value = 0;
        hipMemcpy(deviceOffset, &value, sizeof(unsigned int), hipMemcpyHostToDevice);
        
        hipMemcpy(deviceM, &m, sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMalloc((void**) &deviceMapBabyStep, m * sizeof(ll));
        
        babyStep<<<m, 1>>>(deviceN, deviceM, deviceG, deviceOffset, deviceMapBabyStep);
        hipMemcpy(babyTable[0], deviceMapBabyStep, m * sizeof(ll), hipMemcpyDeviceToHost);
    }
    else
    {
        hipMalloc((void**) &deviceMapBabyStep, BABY_TABLE_COLOUMN_SIZE * sizeof(ll));
        hipMemcpy(deviceM, &BABY_TABLE_COLOUMN_SIZE, sizeof(unsigned int), hipMemcpyHostToDevice);
        
        for (unsigned int i = 0; i < babyTableRowSize - 1; i++)
        {
            hipMemcpy(deviceOffset, &i, sizeof(unsigned int), hipMemcpyHostToDevice);
            babyStep<<<BABY_TABLE_COLOUMN_SIZE, 1>>>(deviceN, deviceM, deviceG, deviceOffset, deviceMapBabyStep);
            hipMemcpy(babyTable[i], deviceMapBabyStep, BABY_TABLE_COLOUMN_SIZE * sizeof(ll), hipMemcpyDeviceToHost);
        }
        
        hipMemcpy(deviceM, &m, sizeof(unsigned int), hipMemcpyHostToDevice);
        unsigned int offset = babyTableRowSize - 1;
        hipMemcpy(deviceOffset, &offset, sizeof(unsigned int), hipMemcpyHostToDevice);
        
        babyStep<<<m, 1>>>(deviceN, deviceM, deviceG, deviceOffset, deviceMapBabyStep);
        hipMemcpy(babyTable[babyTableRowSize - 1], deviceMapBabyStep, m * sizeof(ll), hipMemcpyDeviceToHost);
    }

    if (m <= 100)
    {
        printf("[");
        for (int i = 0; i < babyTableRowSize - 1; i++)
        {
            for (int j = 0; j < BABY_TABLE_COLOUMN_SIZE - 1; j++)
            {
                printf("%llu,", babyTable[i][j]);
            }
        }
        for (int j = 0; j < m; j++)
        {
            printf("%llu,", babyTable[babyTableRowSize - 1][j]);
        }
        printf("\b]\n\n");
    }

    hipFree(deviceN);
    hipFree(deviceM);
    hipFree(deviceG);
    hipFree(deviceMapBabyStep);
    
    for (int i = 0; i < babyTableRowSize; i++) 
    {
        delete [] babyTable[i];
    }

    delete [] babyTable;

    free(mapBabyStep);
}

__global__ void babyStep(const ll *n, const unsigned int *m, const ll *g, const unsigned int *offset, ll *mapBabyStep) 
{
	const unsigned int BABY_TABLE_COLOUMN_SIZE = 65536;
    ll id = blockIdx.x + (BABY_TABLE_COLOUMN_SIZE * *offset);
    cudaPowModll(g, &id, n, &mapBabyStep[id]);
}

__global__ void giantStep(const ll *n, const unsigned int *m, const ll *g, const ll *a, ll *mapBabyStep, unsigned int *resultI, unsigned int *resultJ, int *foundResult)
{
    if (!foundResult)
    {
        ll id = blockIdx.x;
        ll localN, localM;
        localN = *n;
        localM = *m;
        ll exp = (localN - localM - 1) * id;
        ll powResult;
        cudaPowModll(g, &exp, n, &powResult);
        powResult = (powResult * *a) % *n;

        for (unsigned int i = 0; i < *m; i++)
        {
                if (mapBabyStep[i] == powResult && !foundResult)
                {
                    atomicAdd(foundResult, 1);
                    atomicAdd(resultJ, i);
                    atomicAdd(resultI, id);
                    return;
                }
        }
    }
}
typedef struct
{
    ll key;
    ll data;
} CudaPowData;

__device__ void cudaPow(const ll *basis, const ll *exponent, const ll *modulus, ll *result)
{
    ll check1 = 0;
    ll check2 = 1;

    if (*basis == check1)
    {
        *result = check1;
        return;
    }

    if (*exponent == check1)
    {
        *result = check2;
        return;
    }

    if (*exponent == check2)
    {
        *result = *basis;
        return;
    }

    int arraySize = 0;
    int arrayCount = 0;
    getArraySize(exponent, &arraySize);
    
    CudaPowData *values = new CudaPowData[arraySize];
    
    ll globalExp = 1;
    *result = *basis;

    do
    {
        if ((globalExp + globalExp) <= *exponent)
        {
            *result *= *result;
            *result %= *modulus;
            globalExp *= 2;

            CudaPowData data;
            data.key = globalExp;
            data.data = *result;

            values[arrayCount] = data;
        }
        else
        {
            if ((*exponent - globalExp) == 1)
            {
                *result *= *basis;
                *result %= *modulus;
                globalExp += 1;
            }
            else
            {
                for (int i = arraySize - 1; i >= 0; i--)
                {
                    if ((values[i].key + globalExp) <= *exponent)
                    {
                        *result *= values[i].data;
                        *result %= *modulus;
                        globalExp += values[i].key;
                    }
                }
            }
        }
    } while (globalExp != *exponent);

    delete [] values;
}

__device__ void getArraySize(const ll *exp, int *arraySize)
{
    *arraySize = 0;
    ll value = 1;

    while (value <= *exp)
    {
        value *= 2;
        *arraySize++;
    } 
    *arraySize -= 1;
}

__device__ void cudaPowModll(const ll* base, const ll* exp, const ll* mod, ll* result)
{
	if (*exp == 0)
	{
		*result = 1;
		return;
	}

	int i;
	for (i = 62; i>=1; --i)
	{
		if (((*exp>>i)&1) == 1)
		{
			break;
		}
	}
	*result = *base;
	for (--i; i >=0; --i)
	{
		*result *= *result;
		*result %= *mod;
		if ((*exp>>i)&1)
		{
			*result *= *base;
			*result %= *mod;
		}
	}
}
